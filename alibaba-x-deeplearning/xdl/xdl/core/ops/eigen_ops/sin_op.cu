#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct SinFunctor {
  Tout operator()(Tin in) const {
    return sin(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Sin, xdl::SinFunctor)
