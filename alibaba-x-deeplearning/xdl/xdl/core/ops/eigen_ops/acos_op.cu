#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct AcosFunctor {
  Tout operator()(Tin in) const {
    return acos(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Acos, xdl::AcosFunctor)
