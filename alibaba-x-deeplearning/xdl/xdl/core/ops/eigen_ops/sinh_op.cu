#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct SinhFunctor {
  Tout operator()(Tin in) const {
    return sinh(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Sinh, xdl::SinhFunctor)
