#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct AcoshFunctor {
  Tout operator()(Tin in) const {
    return acosh(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Acosh, xdl::AcoshFunctor)
