#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct CosFunctor {
  Tout operator()(Tin in) const {
    return cos(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Cos, xdl::CosFunctor)
