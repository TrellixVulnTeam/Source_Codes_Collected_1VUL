#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct AtanFunctor {
  Tout operator()(Tin in) const {
    return atan(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Atan, xdl::AtanFunctor)
