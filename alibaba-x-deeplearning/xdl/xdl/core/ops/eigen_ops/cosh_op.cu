#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct CoshFunctor {
  Tout operator()(Tin in) const {
    return cosh(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Cosh, xdl::CoshFunctor)
