#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct CeilFunctor {
  Tout operator()(Tin in) const {
    return ceil(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Ceil, xdl::CeilFunctor)
