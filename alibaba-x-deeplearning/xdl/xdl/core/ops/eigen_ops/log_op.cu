#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct LogFunctor {
  Tout operator()(Tin in) const {
    return log(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Log, xdl::LogFunctor)
