#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct SqrtFunctor {
  Tout operator()(Tin in) const {
    return sqrt(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Sqrt, xdl::SqrtFunctor)
