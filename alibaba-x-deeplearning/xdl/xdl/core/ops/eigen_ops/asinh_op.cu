#include "hip/hip_runtime.h"
#include "xdl/core/ops/eigen_ops/unary_eigen_op_common.h"
#include <cmath>

namespace xdl {

template <typename Device, typename Tin, typename Tout>
struct AsinhFunctor {
  Tout operator()(Tin in) const {
    return asinh(in);
  }
};

}

XDL_REGISTER_FLOAT_CALC_UNARY_EIGEN_OP_SIMPLE(Asinh, xdl::AsinhFunctor)
